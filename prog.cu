#include <iostream>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <algorithm>
#include <cstdlib>
#include <ctime>

// Funcion para imprimir el array, ecibe un array arr[] y su tamaño size. Recorre el array e imprime cada elemento, separado por espacios,
// seguido de un salto de línea al final
void printArray(int arr[], int size) {
    for (int i = 0; i < size; i++) {
        std::cout << arr[i] << " ";
    }
    std::cout << std::endl;
}

// Funcion para generar un array aleatorio, utilizando time como semilla generadora de numeros aleatorios y rand() que genera numeros aleatorios
// entre 0 y 999
void generateRandomArray(int* arr, int size) {
    srand(time(NULL));
    for (int i = 0; i < size; i++) {
        arr[i] = rand() % 1000;
    }
}

// FUncion auxiliar de Quicksor, toma un array, un indice bajo y un indice alto, selecciona un pivote y reorganiza los elementos del array de manera
// que los elementos menores al pivote queden a la izquierda y los mayores a la derecha, por ultimo devuelve el indice de la nueva posicion del pivote
int partition(int arr[], int low, int high) {
    int pivot = arr[high];
    int i = low - 1;
    for (int j = low; j <= high - 1; j++) {
        if (arr[j] < pivot) {
            i++;
            std::swap(arr[i], arr[j]);
        }
    }
    std::swap(arr[i + 1], arr[high]);
    return (i + 1);
}

// Quick Sort paralelo en CPU, si el rango de indices es valido, divide el array recursivamente y ordena las dos mitades de forma paralela usando
// la directiva #pragma omp parallel sections
void quickSortParallel(int arr[], int low, int high) {
    if (low < high) {
        int pi = partition(arr, low, high);

        #pragma omp parallel sections
        {
            #pragma omp section
            {
                quickSortParallel(arr, low, pi - 1);
            }
            #pragma omp section
            {
                quickSortParallel(arr, pi + 1, high);
            }
        }
    }
}

// Kernel para realizar la fusion de dos segmentos de un array, este kernel se ejecuta en paralelo en la GPU, donde cada hilo fusiona dos segmentos
// del array, luego se usa un paso (step) para controlar el tamaño de los bloques a fusionar, la cual se realiza comparando elementos entre los
// dos bloques y colocando los mas pequeños en un array temporal
__global__ void mergeSortKernel(int* d_data, int* d_temp, int size, int step) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int start = idx * step * 2;
    if (start < size) {
        int mid = start + step;
        int end = start + 2 * step;

        // Si el final del array esta fuera de rango
        if (mid > size) mid = size;
        if (end > size) end = size;

        // Realizamos la fusion de dos partes
        int i = start;
        int j = mid;
        int k = start;

        while (i < mid && j < end) {
            if (d_data[i] <= d_data[j]) {
                d_temp[k] = d_data[i];
                i++;
            } else {
                d_temp[k] = d_data[j];
                j++;
            }
            k++;
        }

        // Copiar el resto de los elementos
        while (i < mid) {
            d_temp[k] = d_data[i];
            i++;
            k++;
        }

        while (j < end) {
            d_temp[k] = d_data[j];
            j++;
            k++;
        }

        // Copiar el resultado de vuelta a d_data
        for (int i = start; i < end; i++) {
            d_data[i] = d_temp[i];
        }
    }
}

// Merge Sort en GPU, el array se transfiere desde la memoria del host a la memoria del la GPU, para luego ejecutar el kernel de fusion en bloques
// de hilos paralelos, donde a medida que el algoritmo avanza, el tamaño de los segmentos a fusionar crece exponencialmente, asi, al terminar, el
// array ordenado se transfiere de vuelta al host al finalizar
void mergeSort(int* h_data, int size) {
    int* d_data, *d_temp;
    hipMalloc(&d_data, size * sizeof(int));
    hipMalloc(&d_temp, size * sizeof(int));
    hipMemcpy(d_data, h_data, size * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (size + threadsPerBlock - 1) / threadsPerBlock;

    for (int step = 1; step < size; step *= 2) {
        mergeSortKernel<<<blocks, threadsPerBlock>>>(d_data, d_temp, size, step);
        hipDeviceSynchronize();
    }

    hipMemcpy(h_data, d_data, size * sizeof(int), hipMemcpyDeviceToHost);
    hipFree(d_data);
    hipFree(d_temp);
}

// El programa recibe tres parametros, el tamaño del array n, el modo de ejecucion, y el numero de hilos en el caso de la CPU, dependiendo del
// modo, se ejecuta Quick Sort paralelo (CPU) o Merge Sort (GPU), luego se generan datos aleatorios, se ordenan y se imprime el tiempo de ejecucion
// de cada algoritmo.
int main(int argc, char* argv[]) {
    if (argc != 4) {
        std::cout << "Uso: ./prog <n> <modo> <nt>\n";
        return 1;
    }

    int n = std::atoi(argv[1]);  // Tamaño del array
    int mode = std::atoi(argv[2]); // Modo: 0 para CPU, 1 para GPU
    int nt = std::atoi(argv[3]);   // Numero de hilos para OpenMP (modo CPU)

    int* h_data = new int[n];
    int* h_data_copy = new int[n];

    generateRandomArray(h_data, n);
    std::copy(h_data, h_data + n, h_data_copy);

    std::cout << "Array generado: ";
    printArray(h_data, n);

    if (mode == 0) {  // CPU
        // Ordenar usando Quick Sort Paralelo, se configura el numero de hilos para OpenMP
        omp_set_num_threads(nt);
        double start = omp_get_wtime();
        quickSortParallel(h_data, 0, n - 1);
        double end = omp_get_wtime();
        std::cout << "Array ordenado con Quick Sort Paralelo (CPU): ";
        printArray(h_data, n);
        std::cout << "Tiempo de ejecución Quick Sort en CPU: " << (end - start) << " segundos\n";
    } 
    else if (mode == 1) {  // GPU
        // Ordenar usando Merge Sort en GPU
        double start = omp_get_wtime();
        mergeSort(h_data_copy, n);
        double end = omp_get_wtime();
        std::cout << "Array ordenado con Merge Sort Paralelo (GPU): ";
        printArray(h_data_copy, n);
        std::cout << "Tiempo de ejecución Merge Sort en GPU: " << (end - start) << " segundos\n";
    }
    else {
        std::cout << "Modo inválido. Use 0 para CPU o 1 para GPU.\n";
    }

    delete[] h_data;
    delete[] h_data_copy;
    return 0;
}

